#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <assert.h>
#include <fstream>
#include <iomanip>
#include <cstring>
#include "csc.cpp"
#include <malloc.h>
#include <thread>
// use nvprof to caculate time
#define d 0.15
#define blockSize 64
using namespace std;

void thread_fun_read(std::string ,edges& ,int ,std::vector<int>& ,std::vector<int>&);

__global__ void setG_odata(float* g_odata, int N);

__global__ void toleranceReduce(float* d_currentPageRank, float* d_previousPageRank,float* g_odata,int N);

__global__ void change(float* currentPageRank, float* previousPageRank,int N);

__global__ void VertexPush(int* outdegree,int* csc_offset,int* csc_outgoing,const int N, float* currentPageRank, float* previousPageRank);

int main(int argc, char const *argv[]) {
  // read all we need

    clock_t t;
    t = clock();
  edges e;
  edges r_e;
  std::vector<int> row_offset_1,row_offset_2;
  std::vector<int> column_index_1,column_index_2;
  std::thread thread_1(thread_fun_read,argv[1],std::ref(e),0,std::ref(row_offset_1),std::ref(column_index_1));
  std::thread thread_2(thread_fun_read,argv[1],std::ref(r_e),1,std::ref(row_offset_2),std::ref(column_index_2));
  // std::cout<<"hardware_concurrency()"<<std::thread::hardware_concurrency()<<std::endl;
  thread_1.join();
  thread_2.join();
// exit(1);
  unsigned int N_node = row_offset_1.size()-1;
  unsigned int N_edge = column_index_1.size();

  int* outdegree = new int[N_node]();
  int* csc_offset = new int[N_node+1]();
  int* csc_outgoing = new int[N_edge]();

  for(int i = 0;i<N_node;i++){
    outdegree[i] = row_offset_1[i+1]-row_offset_1[i];
  }

  for(int i = 0; i<N_node+1;i++){
    csc_offset[i] = row_offset_1[i];
  }

  for(int i = 0; i<N_edge;i++){
    csc_outgoing[i] = column_index_1[i];
  }

  int N = N_node;

  float* currentPageRank = new float[N] ();
  float* previousPageRank = new float[N] ();
  for(int i = 0; i<N;i++){
    previousPageRank[i] = 1.0/N;
  }
  float* g_odata = new float[1]();

  int* d_outdegree = NULL;
  int* d_csc_offset = NULL;
  int* d_csc_outgoing = NULL;
  float* d_currentPageRank = NULL;
  float* d_previousPageRank = NULL;
  float* d_g_odata = NULL;



  if(hipMalloc(&d_outdegree,N_node*sizeof(int))!=hipSuccess){
    std::cout<<"fail to malloc d_outdegree"<<std::endl;
    return 0;
  };
  if(hipMalloc(&d_csc_offset,(N_node+1)*sizeof(int))!=hipSuccess){
    std::cout<<"fail to malloc d_csc_offset"<<std::endl;
    return 0;
  };
  if(hipMalloc(&d_csc_outgoing,N_edge*sizeof(int))!=hipSuccess){
    std::cout<<"fail to malloc d_csc outgoing"<<std::endl;
    return 0;
  };
  if(hipMalloc(&d_currentPageRank,N*sizeof(float))!=hipSuccess){
    std::cout<<"fail to malloc d_currentPageRank"<<std::endl;
    return 0;
  };
  if(hipMalloc(&d_previousPageRank,N*sizeof(float))!=hipSuccess){
    std::cout<<"fail to malloc d_previousPageRank"<<std::endl;
    return 0;
  };

  if(hipMalloc(&d_g_odata,(N/blockSize+1)*sizeof(float))!=hipSuccess){
    std::cout<<"fail to malloc d_previousPageRank"<<std::endl;
    return 0;
  };

  dim3 threadPerBlock(blockSize);
  dim3 numBlocks((N/threadPerBlock.x+1));

  if( hipMemcpy(d_outdegree,outdegree,N_node*sizeof(int),hipMemcpyHostToDevice) != hipSuccess){
    std::cout<<"fail to hipMemcpy d_outdegree"<<std::endl;
    return 0;
  };

  if( hipMemcpy(d_csc_offset,csc_offset,(N_node+1)*sizeof(int),hipMemcpyHostToDevice) != hipSuccess){
    std::cout<<"fail to hipMemcpy d_csc_offset"<<std::endl;
    return 0;
  };

  if( hipMemcpy(d_csc_outgoing,csc_outgoing,N_edge*sizeof(int),hipMemcpyHostToDevice) != hipSuccess){
    std::cout<<"fail to hipMemcpy d_csc_outgoing"<<std::endl;
    return 0;
  };

  if( hipMemcpy(d_currentPageRank,currentPageRank,N*sizeof(float),hipMemcpyHostToDevice) != hipSuccess){
    std::cout<<"fail to hipMemcpy d_currentPageRank"<<std::endl;
    return 0;
  };
  if( hipMemcpy(d_previousPageRank,previousPageRank,N*sizeof(float),hipMemcpyHostToDevice) != hipSuccess){
    std::cout<<"fail to hipMemcpy d_previousPageRank"<<std::endl;
    return 0;
  };

  int iteration_time = 0;
  int N_g_odata = 1;
  float cc = 0;
  while(iteration_time<2000000){
    cc = 0;
    setG_odata<<<1,1>>>(d_g_odata,1);
    VertexPush<<<numBlocks,threadPerBlock>>>(d_outdegree,d_csc_offset,d_csc_outgoing,N,d_currentPageRank,d_previousPageRank);

    if(hipDeviceSynchronize() != hipSuccess){
      std::cout<<"Synchronize error"<<std::endl;
    };
    for(int i = 0;i<N;i++){
      currentPageRank[i] = previousPageRank[i];
    }

    toleranceReduce<<<1,blockSize,blockSize>>>(d_currentPageRank,d_previousPageRank,d_g_odata,N);

    if( hipMemcpy(g_odata,d_g_odata,sizeof(float),hipMemcpyDeviceToHost) != hipSuccess){

      std::cout<<"fail to hipMemcpy g_odata"<<std::endl;
      return 0;
    };

    for(int i = 0; i<N_g_odata;i++){
      cc += g_odata[i];
    }
    cc = std::sqrt(cc);
    // std::cout<<cc<<std::endl;
    if(cc<10e-7){
      break;
    }

    if(hipDeviceSynchronize() != hipSuccess){
      std::cout<<"Synchronize error"<<std::endl;
    };

    change<<<numBlocks,threadPerBlock>>>(d_currentPageRank,d_previousPageRank,N);
    if(hipDeviceSynchronize() != hipSuccess){
      std::cout<<"Synchronize error"<<std::endl;
    };
    ++iteration_time;
  }

  if( hipMemcpy(previousPageRank,d_currentPageRank,N*sizeof(float),hipMemcpyDeviceToHost) != hipSuccess){

    std::cout<<"fail to hipMemcpy previousPageRank"<<std::endl;
    return 0;
  };

  hipFree(d_outdegree);
  hipFree(d_csc_offset);
  hipFree(d_csc_outgoing);
  hipFree(d_currentPageRank);
  hipFree(d_previousPageRank);
  hipFree(d_g_odata);

  t = clock() -t;
  printf ("%f\n",((float)t)/CLOCKS_PER_SEC);

  std::fstream output;
  output.open("VertexPushResult.txt",std::ios::out);
  output<<std::setprecision(15);
  double sum = 0;
  for(int j = 0;j<N;++j){
    output<<previousPageRank[j]<<std::endl;
    sum +=previousPageRank[j];
  }
  // std::cout<<"sum:"<<sum<<std::endl;
  output.close();
  // t = clock() -t;
  // printf ("%f\n",((float)t)/CLOCKS_PER_SEC);
  return 0;
}

__global__ void VertexPush(int* outdegree,int* csc_offset,int* csc_outgoing,const int N, float* currentPageRank, float* previousPageRank){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N){

    atomicAdd(&currentPageRank[i],d/N);
    float pagerank_value = (1-d)*previousPageRank[i]/outdegree[i];
    for(int j = csc_offset[i];j<csc_offset[i+1];j++){
      atomicAdd(&currentPageRank[csc_outgoing[j]],pagerank_value) ;
    }
  }
}

__global__ void change(float* currentPageRank, float* previousPageRank, int N){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N){
    previousPageRank[i] = currentPageRank[i];
    currentPageRank[i] = 0;
  }
}

__global__ void setG_odata(float* g_odata, int N){
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N){
    g_odata[i] = 0.0;
  }
}

__global__ void toleranceReduce(float* d_currentPageRank, float* d_previousPageRank,float* g_odata,int N)
{
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + tid;
    sdata[tid] = 0;

    while (i < N)
    {
        sdata[tid] += pow((d_currentPageRank[i]-d_previousPageRank[i]),2) + pow((d_currentPageRank[i+blockSize]-d_previousPageRank[i+blockSize]),2);
        i += blockSize*2;
    }

    __syncthreads();
    if (blockSize >= 1024)
    { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
    if (blockSize >= 512)
    { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256)
    { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128)
    { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32)
    {
        if (blockSize >= 64)
        sdata[tid] += sdata[tid + 32];
        __syncthreads();
        if (blockSize >= 32)
        sdata[tid] += sdata[tid + 16];
        __syncthreads();
        if (blockSize >= 16)
        sdata[tid] += sdata[tid + 8];
        __syncthreads();
        if (blockSize >= 8)
        sdata[tid] += sdata[tid + 4];
        __syncthreads();
        if (blockSize >= 4)
        sdata[tid] += sdata[tid + 2];
        __syncthreads();
        if (blockSize >= 2)
        sdata[tid] += sdata[tid + 1];
        __syncthreads();
    }

    if (tid == 0) {
      g_odata[0] = sdata[0];
      // printf("%f\n", sdata[0]);
    }
}

void thread_fun_read(std::string s, edges& e,int flag,std::vector<int>& row_offset,std::vector<int>& column_index){
 graphInfo g = read(s,e,flag);
 csc* ptr = new csc(e,g);

 row_offset = ptr->row_offset;
 column_index = ptr->column_index;

 delete ptr;
}
