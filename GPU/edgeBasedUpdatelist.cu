#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <assert.h>
#include <fstream>
#include <iomanip>
#include <cstring>
#include "csc.cpp"
#include <malloc.h>
#include <thread>
// use nvprof to caculate time
#define d 0.15
#define blockSize 64
using namespace std;

void thread_fun_read(std::string ,edges& ,int ,std::vector<int>& ,std::vector<int>&);

__global__ void setG_odata(float* g_odata, int N);

__global__ void toleranceReduce(float* d_currentPageRank, float* d_previousPageRank,float* g_odata,int N);

__global__ void change(float* currentPageRank, float* previousPageRank,int N);

__global__ void EdgeBaseUpdate(int* outdegree,int* d_col_1,int* d_col_2,const int N,const int offset, float* currentPageRank, float* previousPageRank,const int count);

__global__ void addDampingFactor(const int N, float* currentPageRank);


int main(int argc, char const *argv[]) {
  // read all we need
  clock_t t;
  t = clock();
  edges e;
  edges r_e;
  std::vector<int> row_offset_1,row_offset_2;
  std::vector<int> column_index_1,column_index_2;
  std::thread thread_1(thread_fun_read,argv[1],std::ref(e),0,std::ref(row_offset_1),std::ref(column_index_1));
  std::thread thread_2(thread_fun_read,argv[1],std::ref(r_e),1,std::ref(row_offset_2),std::ref(column_index_2));
  // std::cout<<"hardware_concurrency()"<<std::thread::hardware_concurrency()<<std::endl;
  thread_1.join();
  thread_2.join();
// exit(1);
  unsigned int N_node = row_offset_1.size()-1;
  unsigned int N_edge = column_index_1.size();

  std::vector<int> col1;
  std::vector<int> col2;
  edges e3;
  read2(argv[1],e3,0,col1,col2);

  int* outdegree = new int[N_node]();
  int* csc_offset = new int[N_node+1]();
  int* csc_outgoing = new int[N_edge]();
  int* col_1 = new int[N_edge]();
  int* col_2 = new int[N_edge]();

  for(int i = 0;i<N_node;i++){
    outdegree[i] = row_offset_1[i+1]-row_offset_1[i];
  }
  // printf("%d\n", N_node);
  // exit(1);

  for(int i = 0; i<N_node+1;i++){
    csc_offset[i] = row_offset_1[i];
  }

  for(int i = 0; i<N_edge;i++){
    csc_outgoing[i] = column_index_1[i];
    col_1[i] = col1[i];
    col_2[i] = col2[i];
  }

  int N = N_node;

  float* currentPageRank = new float[N] ();
  float* previousPageRank = new float[N] ();
  for(int i = 0; i<N;i++){
    previousPageRank[i] = 1.0/N;
  }
  float* g_odata = new float[1]();

  int* d_outdegree = NULL;
  int* d_csc_offset = NULL;
  int* d_csc_outgoing = NULL;
  float* d_currentPageRank = NULL;
  float* d_previousPageRank = NULL;
  float* d_g_odata = NULL;
  int* d_col_1 = NULL;
  int* d_col_2 = NULL;



  if(hipMalloc(&d_outdegree,N_node*sizeof(int))!=hipSuccess){
    std::cout<<"fail to malloc d_outdegree"<<std::endl;
    return 0;
  };
  if(hipMalloc(&d_csc_offset,(N_node+1)*sizeof(int))!=hipSuccess){
    std::cout<<"fail to malloc d_csc_offset"<<std::endl;
    return 0;
  };
  if(hipMalloc(&d_csc_outgoing,N_edge*sizeof(int))!=hipSuccess){
    std::cout<<"fail to malloc d_csc outgoing"<<std::endl;
    return 0;
  };
  if(hipMalloc(&d_currentPageRank,N*sizeof(float))!=hipSuccess){
    std::cout<<"fail to malloc d_currentPageRank"<<std::endl;
    return 0;
  };
  if(hipMalloc(&d_previousPageRank,N*sizeof(float))!=hipSuccess){
    std::cout<<"fail to malloc d_previousPageRank"<<std::endl;
    return 0;
  };

  if(hipMalloc(&d_g_odata,(N/blockSize+1)*sizeof(float))!=hipSuccess){
    std::cout<<"fail to malloc d_previousPageRank"<<std::endl;
    return 0;
  };
  if(hipMalloc(&d_col_1,N_edge*sizeof(int))!=hipSuccess){
    std::cout<<"fail to malloc d_col1"<<std::endl;
    return 0;
  };
  if(hipMalloc(&d_col_2,N_edge*sizeof(int))!=hipSuccess){
    std::cout<<"fail to malloc d_col2"<<std::endl;
    return 0;
  };

  // dim3 threadPerBlock(32,32);
  // int dimention = (N_edge/(threadPerBlock.x*threadPerBlock.y)+1);
  // dim3 numBlocks(dimention/512+1,512);
  if( hipMemcpy(d_col_1,col_1,N_edge*sizeof(int),hipMemcpyHostToDevice) != hipSuccess){
    std::cout<<"fail to hipMemcpy d_col_1"<<std::endl;
    return 0;
  };
  if( hipMemcpy(d_col_2,col_2,N_edge*sizeof(int),hipMemcpyHostToDevice) != hipSuccess){
    std::cout<<"fail to hipMemcpy d_col_2"<<std::endl;
    return 0;
  };

  if( hipMemcpy(d_outdegree,outdegree,N_node*sizeof(int),hipMemcpyHostToDevice) != hipSuccess){
    std::cout<<"fail to hipMemcpy d_outdegree"<<std::endl;
    return 0;
  };

  if( hipMemcpy(d_csc_offset,csc_offset,(N_node+1)*sizeof(int),hipMemcpyHostToDevice) != hipSuccess){
    std::cout<<"fail to hipMemcpy d_csc_offset"<<std::endl;
    return 0;
  };

  if( hipMemcpy(d_csc_outgoing,csc_outgoing,N_edge*sizeof(int),hipMemcpyHostToDevice) != hipSuccess){
    std::cout<<"fail to hipMemcpy d_csc_outgoing"<<std::endl;
    return 0;
  };

  if( hipMemcpy(d_currentPageRank,currentPageRank,N*sizeof(float),hipMemcpyHostToDevice) != hipSuccess){
    std::cout<<"fail to hipMemcpy d_currentPageRank"<<std::endl;
    return 0;
  };
  if( hipMemcpy(d_previousPageRank,previousPageRank,N*sizeof(float),hipMemcpyHostToDevice) != hipSuccess){
    std::cout<<"fail to hipMemcpy d_previousPageRank"<<std::endl;
    return 0;
  };

  int iteration_time = 0;
  int N_g_odata = 1;
  float cc = 0;
  dim3 threadPerBlock(1024);
  dim3 numBlocks(1000000/1024+1);
  while(iteration_time<2000000){
    cc = 0;
    setG_odata<<<1,1>>>(d_g_odata,1);
    addDampingFactor<<<N/512+1,512>>>(N,d_currentPageRank);
    if(hipDeviceSynchronize() != hipSuccess){
      std::cout<<"Synchronize error"<<std::endl;
    };
    int calTime = 0;
    while(calTime < N_edge/1000000){
      EdgeBaseUpdate<<<numBlocks,threadPerBlock>>>(d_outdegree,d_col_1,d_col_2,N,calTime,d_currentPageRank,d_previousPageRank,1000000);
      if(hipDeviceSynchronize() != hipSuccess){
        std::cout<<"Synchronize error"<<std::endl;
      };
      calTime++;
    }
    int remaining_edge = N_edge%1000000;
    if(remaining_edge != 0){
      EdgeBaseUpdate<<<remaining_edge/1024+1,1024>>>(d_outdegree,d_col_1,d_col_2,N,calTime,d_currentPageRank,d_previousPageRank,remaining_edge);
      if(hipDeviceSynchronize() != hipSuccess){
        std::cout<<"Synchronize error"<<std::endl;
      };
    }
        // exit(1);
    for(int i = 0;i<N;i++){
      currentPageRank[i] = previousPageRank[i];
    }

    toleranceReduce<<<1,blockSize,blockSize>>>(d_currentPageRank,d_previousPageRank,d_g_odata,N);

    if( hipMemcpy(g_odata,d_g_odata,sizeof(float),hipMemcpyDeviceToHost) != hipSuccess){

      std::cout<<"fail to hipMemcpy g_odata"<<std::endl;
      return 0;
    };

    for(int i = 0; i<N_g_odata;i++){
      cc += g_odata[i];
    }
    cc = std::sqrt(cc);
    // std::cout<<cc<<std::endl;
    if(cc<10e-7){
      break;
    }

    if(hipDeviceSynchronize() != hipSuccess){
      std::cout<<"Synchronize error"<<std::endl;
    };

    change<<<N/512+1,512>>>(d_currentPageRank,d_previousPageRank,N);
    if(hipDeviceSynchronize() != hipSuccess){
      std::cout<<"Synchronize error"<<std::endl;
    };
    ++iteration_time;
  }

  if( hipMemcpy(previousPageRank,d_currentPageRank,N*sizeof(float),hipMemcpyDeviceToHost) != hipSuccess){

    std::cout<<"fail to hipMemcpy previousPageRank"<<std::endl;
    return 0;
  };

  hipFree(d_outdegree);
  hipFree(d_csc_offset);
  hipFree(d_csc_outgoing);
  hipFree(d_currentPageRank);
  hipFree(d_previousPageRank);
  hipFree(d_g_odata);
  hipFree(d_col_1);
  hipFree(d_col_2);

  t = clock() -t;
  printf ("%f\n",((float)t)/CLOCKS_PER_SEC);

  std::fstream output;
  output.open("EdgeBaseUpdatelistResult.txt",std::ios::out);
  output<<std::setprecision(15);
  double sum = 0;
  for(int j = 0;j<N;++j){
    output<<previousPageRank[j]<<std::endl;
    sum +=previousPageRank[j];
  }
  // std::cout<<"sum:"<<sum<<std::endl;
  output.close();
  // t = clock() -t;
  // printf ("%f\n",((float)t)/CLOCKS_PER_SEC);
  return 0;
}

__global__ void addDampingFactor(const int N, float* currentPageRank){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N){
    atomicAdd(&currentPageRank[i],0.15/N);

  }
}

__global__ void EdgeBaseUpdate(int* outdegree,int* d_col_1,int* d_col_2,const int N, const int offset, float* currentPageRank, float* previousPageRank,const int count){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  // int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  // int i = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
  if(i<count){
    int outgoing = d_col_2[i+1000000*offset];
    int origin = d_col_1[i+1000000*offset];
    // int l = 0;
    // int r = N;
    // while(l < r){
    //   int mid = l + (r-l)/2;
    //   if(csc_offset[mid] <= i){
    //     l = mid + 1;
    //   }
    //   else{
    //     r = mid;
    //   }
    // }
    // int origin = l-1;


    float outgoingRank = 0.85*previousPageRank[origin]/outdegree[origin];
    atomicAdd(&currentPageRank[outgoing],outgoingRank);
  }
}

__global__ void change(float* currentPageRank, float* previousPageRank, int N){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N){
    previousPageRank[i] = currentPageRank[i];
    currentPageRank[i] = 0;
  }
}

__global__ void setG_odata(float* g_odata, int N){
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N){
    g_odata[i] = 0.0;
  }
}

__global__ void toleranceReduce(float* d_currentPageRank, float* d_previousPageRank,float* g_odata,int N)
{
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + tid;
    sdata[tid] = 0;

    while (i < N)
    {
        sdata[tid] += pow((d_currentPageRank[i]-d_previousPageRank[i]),2) + pow((d_currentPageRank[i+blockSize]-d_previousPageRank[i+blockSize]),2);
        i += blockSize*2;
    }

    __syncthreads();
    if (blockSize >= 1024)
    { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
    if (blockSize >= 512)
    { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256)
    { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128)
    { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32)
    {
        if (blockSize >= 64)
        sdata[tid] += sdata[tid + 32];
        __syncthreads();
        if (blockSize >= 32)
        sdata[tid] += sdata[tid + 16];
        __syncthreads();
        if (blockSize >= 16)
        sdata[tid] += sdata[tid + 8];
        __syncthreads();
        if (blockSize >= 8)
        sdata[tid] += sdata[tid + 4];
        __syncthreads();
        if (blockSize >= 4)
        sdata[tid] += sdata[tid + 2];
        __syncthreads();
        if (blockSize >= 2)
        sdata[tid] += sdata[tid + 1];
        __syncthreads();
    }

    if (tid == 0) {
      g_odata[0] = sdata[0];
      // printf("%f\n", sdata[0]);
    }
}

void thread_fun_read(std::string s, edges& e,int flag,std::vector<int>& row_offset,std::vector<int>& column_index){
 graphInfo g = read(s,e,flag);
 csc* ptr = new csc(e,g);

 row_offset = ptr->row_offset;
 column_index = ptr->column_index;

 delete ptr;
}
